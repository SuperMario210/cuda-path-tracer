#include "hip/hip_runtime.h"
#include "camera.cuh"
#include "render.cuh"

#define IMPORTANCE_SAMPLING
//#define RUSSIAN_ROULETTE
#define MIN_DEPTH               3
#define MAX_DEPTH               16

__global__ void intersect_scene(BVH *bvh, PathData *paths)
{
    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= MAX_PATHS || !paths->get_flag(index, IS_ACTIVE)) return;

    // Load ray data
    const float3 o = make_float3(paths->origin[index]);
    const float3 d = make_float3(paths->direction[index]);
    float t_max = FLT_MAX;
    float3 n;
    float4 mat;
    uint mat_type;

    // Intersect planes
//    const float3 position = make_float3(0, -0.283, 0);
    const float3 position = make_float3(0, 0, 0);
    const float3 normal = make_float3(0, 1, 0);
    const Plane plane(position, normal, make_float4(0.325, 0.3, 0.35, 0), IS_DIFFUSE);
    plane.intersect(o, d, t_max, n, mat, mat_type);

    // Intersect spheres
    const Sphere sphere1(make_float3(2.025, 0.5, 0), 0.5, make_float4(0.15, 0.25, 0.4, 0.05), IS_GLOSSY);
    const Sphere sphere2(make_float3(0.675, 0.5, 0), 0.5, make_float4(0.5, 0.5, 0.5, 0), IS_MIRROR);
    const Sphere sphere3(make_float3(-0.675, 0.5, 0), 0.5, make_float4(0.7, 0.7, 0.7, 0), IS_DIFFUSE);
    const Sphere sphere4(make_float3(-2.025, 0.5, 0), 0.5, make_float4(1, 1, 1, 1.5), IS_GLASS);
    sphere1.intersect(o, d, t_max, n, mat, mat_type);
    sphere2.intersect(o, d, t_max, n, mat, mat_type);
    sphere3.intersect(o, d, t_max, n, mat, mat_type);
    sphere4.intersect(o, d, t_max, n, mat, mat_type);

    // Intersect triangles
//    bvh->intersect(o, d, t_max, n, mat, mat_type);

    paths->direction[index].w = t_max;
    paths->normal[index] = make_float4(normalize(n));
    paths->material[index] = mat;
    paths->flags[index] = mat_type | IS_ACTIVE;
}

__device__ bool g_is_working = false;

__global__ void logic_kernel(PathData *paths, EnvironmentMap *envmap, float3 *image_data, uint samples_per_pixel, int seed)
{
    uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= MAX_PATHS || !paths->get_flag(index, IS_ACTIVE)) return;

//    paths->flags[index] = 0;
    g_is_working = true;

    hiprandState rand_state;
    hiprand_init(seed + index, 0, 0, &rand_state);

    // Background intersection
    if (paths->direction[index].w == FLT_MAX || paths->depth[index]++ >= MAX_DEPTH) {
        uint pixel_index = paths->pixel_index[index];
        float3 color = make_float3(paths->throughput[index]) * envmap->sample(make_float3(paths->direction[index])) / samples_per_pixel;
        atomicAdd(&image_data[pixel_index].x, color.x);
        atomicAdd(&image_data[pixel_index].y, color.y);
        atomicAdd(&image_data[pixel_index].z, color.z);

        paths->flags[index] = IS_NEW_PATH;
        return;
    }

    // Material Intersection
//    if (paths->get_flag(index, IS_DIFFUSE)) {
//
//    }

#ifdef RUSSIAN_ROULETTE

    if (paths->depth[index]++ >= MIN_DEPTH) {
        float4 throughput = paths->throughput[index];
        float p = fmaxf(throughput.x, fmaxf(throughput.y, throughput.z));
        if (hiprand_uniform(&rand_state) > p) {
            paths->flags[index] = IS_NEW_PATH;
            return;
        }
        paths->throughput[index] *= 1 / p;
    }

#endif
}

__device__ uint g_path_count = 0;

__global__ void generate_primary_paths(Camera *camera, uint width, uint height, uint samples_per_pixel, uint path_count,
                                       PathData *paths, int seed, bool override)
{
    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
//    if (index >= MAX_PATHS || !paths->get_flag(index, IS_NEW_PATH)) return;

    if (index >= MAX_PATHS || (!override && !paths->get_flag(index, IS_NEW_PATH))) return;

    const uint global_index = atomicAdd(&g_path_count, 1) / samples_per_pixel;
    if (global_index >= width * height) return;

    hiprandState rand_state;
    hiprand_init(seed + index, 0, 0, &rand_state);

    const uint x = (global_index) % width;
    const uint y = (global_index / width) % height;

    const float u = (float(x) + hiprand_uniform(&rand_state) - .5f) / float(width - 1);
    const float v = (float(y) + hiprand_uniform(&rand_state) - .5f) / float(height - 1);

    Ray r = camera->cast_ray(u, v, rand_state);
    paths->pixel_index[index] = (height - y - 1) * width + x;
    paths->origin[index] = make_float4(r.origin, EPSILON);
    paths->direction[index] = make_float4(r.direction, FLT_MAX);
    paths->throughput[index] = make_float4(1);
    paths->depth[index] = 0;
    paths->set_flag(index, IS_ACTIVE);
}

__global__ void generate_diffuse_paths(EnvironmentMap *envmap, PathData *paths, int seed)
{
    uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= MAX_PATHS || !paths->get_flag(index, IS_DIFFUSE)) return;

    hiprandState rand_state;
    hiprand_init(seed + index, 0, 0, &rand_state);

    float3 norm = make_float3(paths->normal[index]);
    float3 out_dir = (hiprand_uniform(&rand_state) < 0.5) ? diffuse(norm, rand_state) : envmap->sample_lights(rand_state);

    float4 dir = paths->direction[index];
    dir *= dir.w;
    dir.w = 0;

    paths->origin[index] += dir;
    paths->direction[index] = make_float4(out_dir, FLT_MAX);

    float env_pdf = envmap->pdf(out_dir);
    float diff_pdf = max(dot(norm, out_dir) / PI, 0.0f);
    float mixed_pdf = (env_pdf + diff_pdf) * 0.5f;
    paths->throughput[index] *= paths->material[index] * (diff_pdf / mixed_pdf);
    paths->set_flag(index, IS_ACTIVE);
}

__global__ void generate_glossy_paths(EnvironmentMap *envmap, PathData *paths, int seed)
{
    uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= MAX_PATHS || !paths->get_flag(index, IS_GLOSSY)) return;

    hiprandState rand_state;
    hiprand_init(seed + index, 0, 0, &rand_state);

    float4 dir = paths->direction[index];
    float3 norm = make_float3(paths->normal[index]);
    float4 offset = dir * dir.w;
    offset.w = 0;
    paths->origin[index] += offset;

    if (hiprand_uniform(&rand_state) < 0.05) {
        float3 out_dir = reflect(make_float3(dir), norm);
        paths->direction[index] = make_float4(out_dir, FLT_MAX);
        return;
    }

    float3 out_dir = (hiprand_uniform(&rand_state) < 0.5) ? diffuse(norm, rand_state) : envmap->sample_lights(rand_state);

    paths->direction[index] = make_float4(out_dir, FLT_MAX);

    float env_pdf = envmap->pdf(out_dir);
    float diff_pdf = max(dot(norm, out_dir) / PI, 0.0f);
    float mixed_pdf = (env_pdf + diff_pdf) * 0.5f;
    paths->throughput[index] *= paths->material[index] * (diff_pdf / mixed_pdf);
    paths->set_flag(index, IS_ACTIVE);
}

__global__ void generate_mirror_paths(PathData *paths)
{
    uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= MAX_PATHS || !paths->get_flag(index, IS_MIRROR)) return;

    float4 dir = paths->direction[index];
    float3 norm = make_float3(paths->normal[index]);
    float3 out_dir = reflect(make_float3(dir), norm);

    dir *= dir.w;
    dir.w = 0;

    paths->origin[index] += dir;
    paths->direction[index] = make_float4(out_dir, FLT_MAX);
    paths->throughput[index] *= paths->material[index];
    paths->set_flag(index, IS_ACTIVE);
}

__global__ void generate_glass_paths(PathData *paths, int seed)
{
    uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= MAX_PATHS || !paths->get_flag(index, IS_GLASS)) return;

    hiprandState rand_state;
    hiprand_init(seed + index, 0, 0, &rand_state);

    float3 dir = make_float3(paths->direction[index]);
    float3 norm = make_float3(paths->normal[index]);
    float3 offset = dir * paths->direction[index].w;
    paths->origin[index] += make_float4(offset);

    bool external = dot(dir, norm) < 0;
    norm = external ? norm : -norm;
    float ref_idx = external ? (1 / paths->material[index].w) : paths->material[index].w;
    float cos_t = fmin(dot(-dir, norm), 1.0f);
    float sin_t = sqrtf(1.0f - cos_t * cos_t);
    float3 out_dir;

    if (ref_idx * sin_t > 1.0f || reflectance(cos_t, ref_idx) > hiprand_uniform(&rand_state)) {
        out_dir = reflect(dir, norm);
    } else {
        out_dir = refract(dir, norm, ref_idx);
    }

    paths->direction[index] = make_float4(out_dir, FLT_MAX);
    paths->throughput[index] *= paths->material[index];
    paths->set_flag(index, IS_ACTIVE);
}

__host__ void launch_render_kernel(BVH *bvh, EnvironmentMap *envmap, Camera *camera, float3 *image_data, size_t width,
                                   size_t height, size_t samples_per_pixel, dim3 grid, dim3 block, PathData *paths)
{
    const uint block_size = 128;
    const uint grid_size = (MAX_PATHS + block_size - 1) / block_size;

    uint path_count = 0;
    bool is_working = false;
    bool override = true;
    do {
        generate_primary_paths<<<grid_size, block_size>>>(camera, width, height, samples_per_pixel,
                                                          path_count, paths, rand(), override);

        generate_diffuse_paths<<<grid_size, block_size>>>(envmap, paths, rand());
        generate_mirror_paths<<<grid_size, block_size>>>(paths);
        generate_glossy_paths<<<grid_size, block_size>>>(envmap, paths, rand());
        generate_glass_paths<<<grid_size, block_size>>>(paths, rand());

        intersect_scene<<<grid_size, block_size>>>(bvh, paths);

        logic_kernel<<<grid_size, block_size>>>(paths, envmap, image_data, samples_per_pixel, rand());
        override = false;

        bool temp = false;
        gpuErrchk(hipMemcpyFromSymbol(&is_working, HIP_SYMBOL(g_is_working), sizeof(bool)));
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_is_working), &temp, sizeof(bool)));
    } while (is_working);
}