#include "hip/hip_runtime.h"
#include "camera.cuh"
#include "render.cuh"

#define IMPORTANCE_SAMPLING
#define MAX_DEPTH               16
#define STACK_SIZE              64
#define ENTRYPOINT_SENTINEL     0x76543210
#define FULL_MASK               0xffffffff

__device__ __inline__ int   min_min(int a, int b, int c) { int v; asm("vmin.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   min_max(int a, int b, int c) { int v; asm("vmin.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_min(int a, int b, int c) { int v; asm("vmax.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_max(int a, int b, int c) { int v; asm("vmax.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ float fmin_fmin(float a, float b, float c) { return __int_as_float(min_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmin_fmax(float a, float b, float c) { return __int_as_float(min_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmin(float a, float b, float c) { return __int_as_float(max_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmax(float a, float b, float c) { return __int_as_float(max_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }

__device__ __inline__ float spanBeginKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d){ return fmax_fmax(fminf(a0, a1), fminf(b0, b1), fmin_fmax(c0, c1, d)); }
__device__ __inline__ float spanEndKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d)	{ return fmin_fmin(fmaxf(a0, a1), fmaxf(b0, b1), fmax_fmin(c0, c1, d)); }
__device__ __inline__ void swap(int& a, int& b){ int temp = a; a = b; b = temp;}

__device__ bool intersect_bvh(BVH *bvh, const Ray &r, Intersection &intersect) {
    ///////////////////////////////////////////
    //// KEPLER KERNEL
    ///////////////////////////////////////////

    // BVH layout Compact2 for Kepler
    int traversalStack[STACK_SIZE];

    // Live state during traversal, stored in registers.
    float   origx, origy, origz;    // Ray origin.
    float   dirx, diry, dirz;       // Ray direction.
    float   tmin;                   // t-value from which the ray starts. Usually 0.
    float   idirx, idiry, idirz;    // 1 / ray direction
    float   oodx, oody, oodz;       // ray origin / ray direction

    int*    stackPtr;               // Current position in traversal stack.
    int     leafAddr;               // If negative, then first postponed leaf, non-negative if no leaf (innernode).
    int     nodeAddr;
    int     hitIndex;               // Triangle index of the closest intersection, -1 if none.
    float   hitT;                   // t-value of the closest intersection.
    float3  hitNormal;              // normal of the closest intersection.


    // Initialize (stores local variables in registers)
    {
        origx = r.origin.x;
        origy = r.origin.y;
        origz = r.origin.z;
        dirx = r.direction.x;
        diry = r.direction.y;
        dirz = r.direction.z;
        tmin = EPSILON;

        // ooeps is very small number, used instead of raydir xyz component when that component is near zero
        float ooeps = exp2f(-80.0f); // Avoid div by zero, returns 1/2^80, an extremely small number
        idirx = 1.0f / (fabsf(dirx) > ooeps ? dirx : copysignf(ooeps, dirx)); // inverse ray direction
        idiry = 1.0f / (fabsf(diry) > ooeps ? diry : copysignf(ooeps, diry)); // inverse ray direction
        idirz = 1.0f / (fabsf(dirz) > ooeps ? dirz : copysignf(ooeps, dirz)); // inverse ray direction
        oodx = origx * idirx;  // ray origin / ray direction
        oody = origy * idiry;  // ray origin / ray direction
        oodz = origz * idirz;  // ray origin / ray direction

        // Setup traversal + initialisation

        traversalStack[0] = ENTRYPOINT_SENTINEL; // Bottom-most entry. 0x76543210 (1985229328 in decimal)
        stackPtr = &traversalStack[0]; // point stackPtr to bottom of traversal stack = EntryPointSentinel
        leafAddr = 0;   // No postponed leaf.
        nodeAddr = 0;   // Start from the root.
        hitIndex = -1;  // No triangle intersected so far.
        hitT = intersect.t; // tmax
    }

    // Traversal loop.

    while (nodeAddr != ENTRYPOINT_SENTINEL)
    {
        // Traverse internal nodes until all SIMD lanes have found a leaf.

        bool searchingLeaf = true; // required for warp efficiency
        while (nodeAddr >= 0 && nodeAddr != ENTRYPOINT_SENTINEL)
        {
            // Fetch AABBs of the two child nodes.

            // nodeAddr is an offset in number of bytes (char) in gpuNodes array

            float4 n0xy = tex1Dfetch<float4>(bvh->nodes_texture, nodeAddr * 4); // childnode 0, xy-bounds (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
            float4 n1xy = tex1Dfetch<float4>(bvh->nodes_texture, nodeAddr * 4 + 1); // childnode 1, xy-bounds (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
            float4 nz = tex1Dfetch<float4>(bvh->nodes_texture, nodeAddr * 4 + 2); // childnode 0 and 1, z-bounds (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)
            float4 tmp = tex1Dfetch<float4>(bvh->nodes_texture, nodeAddr * 4 + 3); // contains indices to 2 childnodes in case of innernode, see below
            int2 cnodes = *(int2*)&tmp; // cast first two floats to int
            // (childindex = size of array during building, see CudaBVH.cpp)

            // compute ray intersections with BVH node bounding box

            /// RAY BOX INTERSECTION
            // Intersect the ray against the child nodes.
            float c0lox = n0xy.x * idirx - oodx; // n0xy.x = c0.lo.x, child 0 minbound x
            float c0hix = n0xy.y * idirx - oodx; // n0xy.y = c0.hi.x, child 0 maxbound x
            float c0loy = n0xy.z * idiry - oody; // n0xy.z = c0.lo.y, child 0 minbound y
            float c0hiy = n0xy.w * idiry - oody; // n0xy.w = c0.hi.y, child 0 maxbound y
            float c0loz = nz.x   * idirz - oodz; // nz.x   = c0.lo.z, child 0 minbound z
            float c0hiz = nz.y   * idirz - oodz; // nz.y   = c0.hi.z, child 0 maxbound z
            float c1loz = nz.z   * idirz - oodz; // nz.z   = c1.lo.z, child 1 minbound z
            float c1hiz = nz.w   * idirz - oodz; // nz.w   = c1.hi.z, child 1 maxbound z
            float c0min = spanBeginKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, tmin); // Tesla does max4(min, min, min, tmin)
            float c0max = spanEndKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, hitT); // Tesla does min4(max, max, max, tmax)
            float c1lox = n1xy.x * idirx - oodx; // n1xy.x = c1.lo.x, child 1 minbound x
            float c1hix = n1xy.y * idirx - oodx; // n1xy.y = c1.hi.x, child 1 maxbound x
            float c1loy = n1xy.z * idiry - oody; // n1xy.z = c1.lo.y, child 1 minbound y
            float c1hiy = n1xy.w * idiry - oody; // n1xy.w = c1.hi.y, child 1 maxbound y
            float c1min = spanBeginKepler(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, tmin);
            float c1max = spanEndKepler(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, hitT);

            // ray box intersection boundary tests:
            bool traverseChild0 = (c0min <= c0max);
            bool traverseChild1 = (c1min <= c1max);

            // Neither child was intersected => pop stack.
            if (!traverseChild0 && !traverseChild1) {
                nodeAddr = *stackPtr; // fetch next node by popping the stack
                stackPtr--; // popping decrements stackPtr by 4 bytes (because stackPtr is a pointer to char)
            }

                // Otherwise, one or both children intersected => fetch child pointers.
            else {
                // set nodeAddr equal to intersected childnode index (or first childnode when both children are intersected)
                nodeAddr = (traverseChild0) ? cnodes.x : cnodes.y;

                // Both children were intersected => push the farther one on the stack.
                if (traverseChild0 && traverseChild1) // store closest child in nodeAddr, swap if necessary
                {
                    if (c1min < c0min) swap(nodeAddr, cnodes.y);
                    stackPtr++;
                    *stackPtr = cnodes.y; // push furthest node on the stack
                }
            }

            // First leaf => postpone and continue traversal.
            // leafnodes have a negative index to distinguish them from inner nodes
            // if nodeAddr less than 0 -> nodeAddr is a leaf
            if (nodeAddr < 0 && leafAddr >= 0)
            {
                searchingLeaf = false; // required for warp efficiency
                leafAddr = nodeAddr;
                nodeAddr = *stackPtr;  // pops next node from stack
                stackPtr--;
            }

            // All SIMD lanes have found a leaf => process them.

            // to increase efficiency, check if all the threads in a warp have found a leaf before proceeding to the
            // ray/triangle intersection routine
            if (!__any_sync(FULL_MASK, searchingLeaf))
                break;    // break from while loop and go to code below, processing leaf nodes

        }

        ///////////////////////////////////////////
        /// TRIANGLE INTERSECTION
        //////////////////////////////////////

        // Process postponed leaf nodes.
        while (leafAddr < 0) { /// if leafAddr is negative, it points to an actual leafnode (when positive or 0 it's an innernode)
            // Intersect the ray against each triangle using Sven Woop's algorithm.
            // Woop ray triangle intersection: Woop triangles are unit triangles. Each ray
            // must be transformed to "unit triangle space", before testing for intersection

            // triAddr is index in triWoop array (and bitwise complement of leafAddr)
            for (int triAddr = ~leafAddr;; triAddr += 3) { // no defined upper limit for loop, continues until leaf terminator code 0x80000000 is encountered
                // Read first 16 bytes of the triangle.
                // fetch first precomputed triangle edge
                float4 v00 = bvh->triangles[triAddr];

                // End marker 0x80000000 (negative zero) => all triangles in leaf processed --> terminate
                if (__float_as_int(v00.x) == 0x80000000)
                    break;

                // Compute and check intersection t-value (hit distance along ray).
                float Oz = v00.w - origx*v00.x - origy*v00.y - origz*v00.z;   // Origin z
                float invDz = 1.0f / (dirx*v00.x + diry*v00.y + dirz*v00.z);  // inverse Direction z
                float t = Oz * invDz;

                if (t > tmin && t < hitT) {
                    // Compute and check barycentric u.
                    // fetch second precomputed triangle edge
                    float4 v11 = bvh->triangles[triAddr + 1];
                    float Ox = v11.w + origx*v11.x + origy*v11.y + origz*v11.z;  // Origin.x
                    float Dx = dirx * v11.x + diry * v11.y + dirz * v11.z;  // Direction.x
                    float u = Ox + t * Dx; /// parametric equation of a ray (intersection point)

                    if (u >= 0.0f && u <= 1.0f) {
                        // Compute and check barycentric v.
                        // fetch third precomputed triangle edge
                        float4 v22 = bvh->triangles[triAddr + 2];
                        float Oy = v22.w + origx*v22.x + origy*v22.y + origz*v22.z;
                        float Dy = dirx*v22.x + diry*v22.y + dirz*v22.z;
                        float v = Oy + t*Dy;

                        if (v >= 0.0f && u + v <= 1.0f) {
                            // We've got a hit!
                            // Record intersection.
                            hitT = t;
                            hitIndex = triAddr; // store triangle index for shading

//                            // Closest intersection not required => terminate.
//                            if (anyHit)  // only true for shadow rays
//                            {
//                                nodeAddr = EntrypointSentinel;
//                                break;
//                            }

                            // compute normal vector by taking the cross product of two edge vectors
                            // because of Woop transformation, only one set of vectors works
                            hitNormal = cross(make_float3(v11.x, v11.y, v11.z), make_float3(v22.x, v22.y, v22.z));
                        }
                    }
                }
            } // end triangle intersection

            // Another leaf was postponed => process it as well.
            leafAddr = nodeAddr;
            if (nodeAddr < 0)    // nodeAddr is an actual leaf when < 0
            {
                nodeAddr = *stackPtr;  // pop stack
                stackPtr--;
            }
        } // end leaf/triangle intersection loop
    } // end traversal loop (AABB and triangle intersection)

    // Remap intersected triangle index, and store the result.
    if (hitIndex != -1) {
        intersect.t = hitT;
        intersect.position = r.at(hitT);
        intersect.set_normal(r, normalize(hitNormal));
        return true;
    } else {
        return false;
    }
}

__device__ float3 path_trace(Ray r, hiprandState &rand_state, EnvironmentMap *envmap, BVH *bvh)
{
    Material plane_mat(LAMBERTIAN, make_float3(0.65, 0.1, 0.1), 0);
    Plane plane(make_float3(0, -0.283, 0), make_float3(0, 1, 0), &plane_mat);

//    Material sphere1_mat(GLASS, make_float3(1), 1.5);
//    Sphere sphere1(make_float3(-2.025, 0.5, 0), 0.5, &sphere1_mat);
//
//    Material sphere2_mat(LAMBERTIAN, make_float3(0.7), 0);
//    Sphere sphere2(make_float3(-0.675, 0.5, 0), 0.5, &sphere2_mat);
//
//    Material sphere3_mat(MIRROR, make_float3(0.5), 0);
//    Sphere sphere3(make_float3(0.675, 0.5, 0), 0.5, &sphere3_mat);
//
//    Material sphere4_mat(GLOSSY, make_float3(0.15, 0.25, 0.4), 0.05);
//    Sphere sphere4(make_float3(2.025, 0.5, 0), 0.5, &sphere4_mat);

    Material bvh_mat(LAMBERTIAN, make_float3(0.65, 0.65, 0.65), 0);
//    Material bvh_mat(GLASS, make_float3(1), 1.5);

    float3 color = make_float3(1);
    for (int depth = 0; depth < MAX_DEPTH; depth++) {
        // Intersect scene
        Intersection intersect;
        bool hit0 = plane.intersect(r, intersect);
        bool hit1 = intersect_bvh(bvh, r, intersect);
        if (hit1) {
            intersect.material = &bvh_mat;
        }

//        bool hit1 = sphere1.intersect(r, intersect);
//        bool hit2 = sphere2.intersect(r, intersect);
//        bool hit3 = sphere3.intersect(r, intersect);
//        bool hit4 = sphere4.intersect(r, intersect);
//
//        if (hit0 || hit1 || hit2 || hit3 || hit4) {
        if (hit0 || hit1) {

#ifdef IMPORTANCE_SAMPLING

            float3 attenuation;
            bool importance_sample = false;
            Ray r_new = intersect.material->brdf(r, intersect, attenuation, rand_state, importance_sample);

            if (importance_sample) {
                if (hiprand_uniform(&rand_state) < 0.5) {
                    r_new.direction = envmap->sample_lights(rand_state);
                }

                float env_pdf = envmap->pdf(r_new.direction);
                float diff_pdf = intersect.material->pdf(r, intersect, r_new);
                float mixed_pdf = (env_pdf + diff_pdf) * 0.5f;
                color *= diff_pdf / mixed_pdf;
            }

            color *= attenuation;
            r = r_new;

#else

            float3 attenuation;
            bool importance_sample = false;
            r = intersect.material->brdf(r, intersect, attenuation, rand_state, importance_sample);
            color *= attenuation;

#endif

        } else {
            return color * envmap->sample(r.direction);
        }
    }

    return make_float3(0);
}

__global__ void render_kernel(BVH *bvh, EnvironmentMap *envmap, Camera *camera, float3 *image_data, size_t width,
                              size_t height, size_t samples_per_pixel)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t i = (height - y - 1) * width + x;

    hiprandState rand_state;
    hiprand_init(1984 + i, 0, 0, &rand_state);

    float3 accum_color = make_float3(0);
    for (int samp = 0; samp < samples_per_pixel; samp++) {
        float u = (float(x) + hiprand_uniform(&rand_state) - .5f) / float(width - 1);
        float v = (float(y) + hiprand_uniform(&rand_state) - .5f) / float(height - 1);

        Ray ray = camera->cast_ray(u, v, rand_state);
        accum_color += path_trace(ray, rand_state, envmap, bvh);
    }

    image_data[i] = accum_color / samples_per_pixel;
}

__global__ void intersect_scene(BVH *bvh, PathData *paths)
{
    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= MAX_PATHS || !paths->get_flag(index, IS_ACTIVE)) return;

    // Load ray data
    const uint path_index = index;
    float4 o = paths->origin[path_index];
    float4 d = paths->direction[path_index];
    float3 n;
    float t_min  = o.w;
    float t_max  = d.w;

    // Intersect planes
    const float3 normal = make_float3(0, 1, 0);
    const float3 position = make_float3(0, -0.283, 0);

    float det = dot(make_float3(d), normal);
    if (det > EPSILON || det < -EPSILON) {
        auto t = dot(position - make_float3(o), normal) / det;
        if (t > t_min && t < t_max) {
            t_max = t;
            n = normal;
        }
    }

    // Traversal stack in CUDA thread-local memory.
    int traversalStack[STACK_SIZE];

    // Live state during traversal, stored in registers.
    int stackPtr = 0;
    int node_addr = 0;
    float3 idir = 1.0f / make_float3(d);
    float3 ood  = idir * make_float3(o);

    // Traversal loop.
    while(stackPtr >= 0) {
        while (node_addr >= 0 && stackPtr >= 0) {
            // Fetch AABBs of the two child nodes.
            const float4 n0xy = tex1Dfetch<float4>(bvh->nodes_texture, node_addr * 4 + 0); // (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
            const float4 n1xy = tex1Dfetch<float4>(bvh->nodes_texture, node_addr * 4 + 1); // (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
            const float4 nz   = tex1Dfetch<float4>(bvh->nodes_texture, node_addr * 4 + 2); // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)
            const float4 tmp  = tex1Dfetch<float4>(bvh->nodes_texture, node_addr * 4 + 3); // child_index0, child_index1
            int2 cnodes       = *(int2*)&tmp;

            // Intersect the ray against the child nodes.
            const float c0lox = n0xy.x * idir.x - ood.x;
            const float c0hix = n0xy.y * idir.x - ood.x;
            const float c0loy = n0xy.z * idir.y - ood.y;
            const float c0hiy = n0xy.w * idir.y - ood.y;
            const float c0loz = nz.x   * idir.z - ood.z;
            const float c0hiz = nz.y   * idir.z - ood.z;
            const float c1loz = nz.z   * idir.z - ood.z;
            const float c1hiz = nz.w   * idir.z - ood.z;
            const float c0min = spanBeginKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, t_min);
            const float c0max = spanEndKepler  (c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, t_max);
            const float c1lox = n1xy.x * idir.x - ood.x;
            const float c1hix = n1xy.y * idir.x - ood.x;
            const float c1loy = n1xy.z * idir.y - ood.y;
            const float c1hiy = n1xy.w * idir.y - ood.y;
            const float c1min = spanBeginKepler(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, t_min);
            const float c1max = spanEndKepler  (c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, t_max);

            const bool traverseChild0 = (c0max >= c0min);
            const bool traverseChild1 = (c1max >= c1min);

            // Otherwise => fetch child pointers.
            if (traverseChild0 || traverseChild1) {
                node_addr = (traverseChild0) ? cnodes.x : cnodes.y;

                // Both children were intersected => push the farther one.
                if (traverseChild0 && traverseChild1) {
                    if (c1min < c0min) {
                        swap(node_addr, cnodes.y);
                    }

                    traversalStack[++stackPtr] = cnodes.y;
                }
            } else {
                node_addr = traversalStack[stackPtr--];
            }
        }

        while (node_addr < 0 && stackPtr >= 0) {
            for (int tri_addr = ~node_addr;; tri_addr += 3) {
                const float4 v00 = bvh->triangles[tri_addr + 0];
                const float4 v11 = bvh->triangles[tri_addr + 1];
                const float4 v22 = bvh->triangles[tri_addr + 2];

                // End marker (negative zero) => all triangles processed.
                if (__float_as_int(v00.x) == 0x80000000) {
                    break;
                }

                // Woop triangle intersection
                float Oz = v00.w - o.x*v00.x - o.y*v00.y - o.z*v00.z;
                float invDz = 1.0f / (d.x*v00.x + d.y*v00.y + d.z*v00.z);
                float t = Oz * invDz;

                if (t > t_min && t < t_max) {
                    // Compute and check barycentric u.
                    float Ox = v11.w + o.x*v11.x + o.y*v11.y + o.z*v11.z;
                    float Dx = d.x*v11.x + d.y*v11.y + d.z*v11.z;
                    float u = Ox + t*Dx;

                    if (u >= 0.0f) {
                        // Compute and check barycentric v.
                        float Oy = v22.w + o.x*v22.x + o.y*v22.y + o.z*v22.z;
                        float Dy = d.x*v22.x + d.y*v22.y + d.z*v22.z;
                        float v = Oy + t*Dy;

                        if (v >= 0.0f && u + v <= 1.0f) {
                            // Record intersection.
                            t_max = t;
                            n = cross(make_float3(v11.x, v11.y, v11.z), make_float3(v22.x, v22.y, v22.z));
                        }
                    }
                }
            } // triangle

            node_addr = traversalStack[stackPtr--];
        } // leaf
    } // traversal

    paths->direction[path_index].w = t_max;
    paths->normal[path_index] = make_float4(normalize(n));
}

__device__ __inline__ float3 diffuse(const float3 &n, hiprandState &rand_state) {
    // randomly generate point in sphere
    float z = hiprand_uniform(&rand_state) * 2.0f - 1.0f;
    float a = hiprand_uniform(&rand_state) * 2.0f * PI;
    float r = sqrtf(1.0f - z * z);
    float x = r * cosf(a);
    float y = r * sinf(a);
    float3 dir = make_float3(x, y, z) + n;

    if (dir.x < EPSILON && dir.y < EPSILON && dir.z < EPSILON)
        return n;

    return normalize(dir);
}

__device__ bool g_is_working = false;

__global__ void logic_kernel(PathData *paths, EnvironmentMap *envmap, float3 *image_data, uint samples_per_pixel)
{
    uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= MAX_PATHS || !paths->get_flag(index, IS_ACTIVE)) return;

    paths->flags[index] = 0;
    g_is_working = true;

    // TERMINATE PATH
    if (paths->direction[index].w == FLT_MAX || paths->depth[index]++ >= MAX_DEPTH) {
        uint pixel_index = paths->pixel_index[index];
        float3 color = make_float3(paths->throughput[index]) * envmap->sample(make_float3(paths->direction[index])) / samples_per_pixel;
        atomicAdd(&image_data[pixel_index].x, color.x);
        atomicAdd(&image_data[pixel_index].y, color.y);
        atomicAdd(&image_data[pixel_index].z, color.z);

        paths->set_flag(index, IS_NEW_PATH);
    } else {
        // TODO: check actual material
        paths->set_flag(index, IS_DIFFUSE);
    }
}

__device__ uint g_path_count = 0;

__global__ void generate_primary_paths(Camera *camera, uint width, uint height, uint samples_per_pixel, uint path_count,
                                       PathData *paths, int seed, bool override)
{
    const uint index = blockDim.x * blockIdx.x + threadIdx.x;
//    if (index >= MAX_PATHS || !paths->get_flag(index, IS_NEW_PATH)) return;

    if (index >= MAX_PATHS || (!override && !paths->get_flag(index, IS_NEW_PATH))) return;

    const uint global_index = atomicAdd(&g_path_count, 1) / samples_per_pixel;
    if (global_index >= width * height) return;

    hiprandState rand_state;
    hiprand_init(seed + index, 0, 0, &rand_state);

    const uint x = (global_index) % width;
    const uint y = (global_index / width) % height;

    const float u = (float(x) + hiprand_uniform(&rand_state) - .5f) / float(width - 1);
    const float v = (float(y) + hiprand_uniform(&rand_state) - .5f) / float(height - 1);

    Ray r = camera->cast_ray(u, v, rand_state);
    paths->pixel_index[index] = (height - y - 1) * width + x;
    paths->origin[index] = make_float4(r.origin, EPSILON);
    paths->direction[index] = make_float4(r.direction, FLT_MAX);
    paths->throughput[index] = make_float4(1);
    paths->depth[index] = 0;
    paths->set_flag(index, IS_ACTIVE);
}

__global__ void generate_diffuse_paths(EnvironmentMap *envmap, PathData *paths, int seed)
{
    uint index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= MAX_PATHS || !paths->get_flag(index, IS_DIFFUSE)) return;

    hiprandState rand_state;
    hiprand_init(seed + index, 0, 0, &rand_state);

    float3 norm = make_float3(paths->normal[index]);
    float3 out_dir = (hiprand_uniform(&rand_state) < 0.5) ? diffuse(norm, rand_state) : envmap->sample_lights(rand_state);

    float4 dir = paths->direction[index];
    dir *= dir.w;
    dir.w = 0;

    paths->origin[index] += dir;
    paths->direction[index] = make_float4(out_dir, FLT_MAX);

    float env_pdf = envmap->pdf(out_dir);
    float diff_pdf = max(dot(norm, out_dir) / PI, 0.0f);
    float mixed_pdf = (env_pdf + diff_pdf) * 0.5f;
    paths->throughput[index] *= make_float4(0.65f * diff_pdf / mixed_pdf);
    paths->set_flag(index, IS_ACTIVE);
}


__host__ void launch_render_kernel(BVH *bvh, EnvironmentMap *envmap, Camera *camera, float3 *image_data, size_t width,
                                   size_t height, size_t samples_per_pixel, dim3 grid, dim3 block, PathData *paths)
{
//    render_kernel <<< grid, block >>>(bvh, envmap, camera, image_data, width, height, samples_per_pixel);
//    return;

    const uint block_size = 64 * 2;
    const uint grid_size = (MAX_PATHS + block_size - 1) / block_size;

    uint path_count = 0;
    int i = 1021;

    bool is_working = false;
    bool override = true;
    do {
        generate_primary_paths<<<grid_size, block_size>>>(camera, width, height, samples_per_pixel,
                                                          path_count, paths, i++, override);

        generate_diffuse_paths<<<grid_size, block_size>>>(envmap, paths, rand());

        intersect_scene<<<grid_size, block_size>>>(bvh, paths);

        logic_kernel<<<grid_size, block_size>>>(paths, envmap, image_data, samples_per_pixel);
        override = false;

        bool temp = false;
        gpuErrchk(hipMemcpyFromSymbol(&is_working, HIP_SYMBOL(g_is_working), sizeof(bool)));
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_is_working), &temp, sizeof(bool)));
    } while (is_working);
}