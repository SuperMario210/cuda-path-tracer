#include "hip/hip_runtime.h"
#include "camera.cuh"
#include "render.cuh"
#include "material.cuh"
#include "object.cuh"

#define IMPORTANCE_SAMPLING
#define MAX_DEPTH   16


__device__ float4 EnvironmentMap::sample_texture(hipTextureObject_t texture_obj, float u, float v) {
    return tex2D<float4>(texture_obj, u, v);
}

__device__ float3 path_trace(Ray r, hiprandState &rand_state, EnvironmentMap *envmap)
{
    Material plane_mat(LAMBERTIAN, make_float3(0.325, 0.3, 0.35), 0);
    Plane plane(make_float3(0), make_float3(0, 1, 0), &plane_mat);

    Material sphere1_mat(GLASS, make_float3(1), 1.5);
    Sphere sphere1(make_float3(-2.025, 0.5, 0), 0.5, &sphere1_mat);

    Material sphere2_mat(LAMBERTIAN, make_float3(0.7), 0);
    Sphere sphere2(make_float3(-0.675, 0.5, 0), 0.5, &sphere2_mat);

    Material sphere3_mat(MIRROR, make_float3(0.5), 0);
    Sphere sphere3(make_float3(0.675, 0.5, 0), 0.5, &sphere3_mat);

    Material sphere4_mat(GLOSSY, make_float3(0.15, 0.25, 0.4), 0.05);
    Sphere sphere4(make_float3(2.025, 0.5, 0), 0.5, &sphere4_mat);

    float3 color = make_float3(1);
    for (int depth = 0; depth < MAX_DEPTH; depth++) {
        // Intersect scene
        Intersection intersect;
        bool hit0 = plane.intersect(r, intersect);
        bool hit1 = sphere1.intersect(r, intersect);
        bool hit2 = sphere2.intersect(r, intersect);
        bool hit3 = sphere3.intersect(r, intersect);
        bool hit4 = sphere4.intersect(r, intersect);

        if (hit0 || hit1 || hit2 || hit3 || hit4) {

#ifdef IMPORTANCE_SAMPLING

            float3 attenuation;
            bool importance_sample = false;
            Ray r_new = intersect.material->brdf(r, intersect, attenuation, rand_state, importance_sample);

            if (importance_sample) {
                if (hiprand_uniform(&rand_state) < 0.5) {
                    r_new.direction = envmap->sample_lights(rand_state);
                }

                float env_pdf = envmap->pdf(r_new.direction);
                float diff_pdf = intersect.material->pdf(r, intersect, r_new);
                float mixed_pdf = (env_pdf + diff_pdf) * 0.5f;
                color *= diff_pdf / mixed_pdf;
            }

            color *= attenuation;
            r = r_new;

#else

            float3 attenuation;
            bool importance_sample = false;
            r = intersect.material->brdf(r, intersect, attenuation, rand_state, importance_sample);
            color *= attenuation;

#endif

        } else {
            return color * envmap->sample(r.direction);
        }
    }

    return make_float3(0);
}

__global__ void render_kernel(EnvironmentMap *envmap, Camera *camera, float3 *image_data, size_t width, size_t height,
                              size_t samples_per_pixel)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t i = (height - y - 1) * width + x;

    hiprandState rand_state;
    hiprand_init(1984 + i, 0, 0, &rand_state);

    float3 accum_color = make_float3(0);
    for (int samp = 0; samp < samples_per_pixel; samp++) {
        float u = (float(x) + hiprand_uniform(&rand_state) - .5f) / float(width - 1);
        float v = (float(y) + hiprand_uniform(&rand_state) - .5f) / float(height - 1);

        Ray ray = camera->cast_ray(u, v, rand_state);
        accum_color += path_trace(ray, rand_state, envmap);
    }

    image_data[i] = accum_color / samples_per_pixel;
}

__host__ void launch_render_kernel(EnvironmentMap *envmap, Camera *camera, float3 *image_data, size_t width, size_t height,
                                   size_t samples_per_pixel, dim3 grid, dim3 block)
{
    render_kernel <<< grid, block >>>(envmap, camera, image_data, width, height, samples_per_pixel);
}