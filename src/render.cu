#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include "camera.cuh"
#include "object.cuh"
#include "environment_map.cuh"
#include "material.cuh"
#include <hiprand/hiprand_kernel.h>

#define IMPORTANCE_SAMPLING
#define MAX_DEPTH   16


__device__ float3 path_trace(Ray r, hiprandState &rand_state, EnvironmentMap *envmap)
{
    Material plane_mat(LAMBERTIAN, make_float3(0.325, 0.3, 0.35), 0);
    Plane plane(make_float3(0), make_float3(0, 1, 0), &plane_mat);

    Material sphere1_mat(GLASS, make_float3(1), 1.5);
    Sphere sphere1(make_float3(-2.025, 0.5, 0), 0.5, &sphere1_mat);

    Material sphere2_mat(LAMBERTIAN, make_float3(0.7), 0);
    Sphere sphere2(make_float3(-0.675, 0.5, 0), 0.5, &sphere2_mat);

    Material sphere3_mat(MIRROR, make_float3(0.5), 0);
    Sphere sphere3(make_float3(0.675, 0.5, 0), 0.5, &sphere3_mat);

    Material sphere4_mat(GLOSSY, make_float3(0.15, 0.25, 0.4), 0.05);
    Sphere sphere4(make_float3(2.025, 0.5, 0), 0.5, &sphere4_mat);

    float3 color = make_float3(1);
    for (int depth = 0; depth < MAX_DEPTH; depth++) {
        // Intersect scene
        Intersection intersect;
        bool hit0 = plane.intersect(r, intersect);
        bool hit1 = sphere1.intersect(r, intersect);
        bool hit2 = sphere2.intersect(r, intersect);
        bool hit3 = sphere3.intersect(r, intersect);
        bool hit4 = sphere4.intersect(r, intersect);

        if (hit0 || hit1 || hit2 || hit3 || hit4) {

#ifdef IMPORTANCE_SAMPLING

            float3 attenuation;
            bool importance_sample = false;
            Ray r_new = intersect.material->brdf(r, intersect, attenuation, rand_state, importance_sample);

            if (importance_sample) {
                if (hiprand_uniform(&rand_state) < 0.5) {
                    r_new.direction = envmap->sample_lights(rand_state);
                }

                float env_pdf = envmap->pdf(r_new.direction);
                float diff_pdf = intersect.material->pdf(r, intersect, r_new);
                float mixed_pdf = (env_pdf + diff_pdf) * 0.5f;
                color *= diff_pdf / mixed_pdf;
            }

            color *= attenuation;
            r = r_new;

#else

            float3 attenuation;
            bool importance_sample = false;
            r = intersect.material->brdf(r, intersect, attenuation, rand_state, importance_sample);
            color *= attenuation;

#endif

        } else {
            return color * envmap->sample(r.direction);
        }
    }

    return make_float3(0);
}

__global__ void render_kernel(EnvironmentMap *envmap, Camera *camera, float3 *image_data, size_t width, size_t height,
                              size_t samples_per_pixel)
{
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t i = (height - y - 1) * width + x;

    hiprandState rand_state;
    hiprand_init(1984 + i, 0, 0, &rand_state);

    float3 accum_color = make_float3(0);
    for (int samp = 0; samp < samples_per_pixel; samp++) {
        float u = (float(x) + hiprand_uniform(&rand_state) - .5f) / float(width - 1);
        float v = (float(y) + hiprand_uniform(&rand_state) - .5f) / float(height - 1);

        Ray ray = camera->cast_ray(u, v, rand_state);
        accum_color += path_trace(ray, rand_state, envmap);
    }

    image_data[i] = accum_color / samples_per_pixel;
}